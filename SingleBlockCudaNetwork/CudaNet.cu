#include "hip/hip_runtime.h"
#include "SingleBlockCudaNetwork.h"

void CudaNetwork::initialize_batch_data(size_t batch_size) {
	m_batch_activation_size = 0;

	m_batch_data_size = m_network_size;

	for (size_t i = 0; i < m_dimensions.size(); i++) {
		m_batch_data_size += 3 * (m_dimensions[i] * batch_size);
		m_batch_activation_size += m_dimensions[i] * batch_size;
	}

	hipMalloc(&m_batch_data, m_batch_data_size * sizeof(float));

	m_activation = &m_batch_data[m_batch_activation_size];

	m_d_total = &m_activation[m_batch_activation_size];
	m_d_weights = &m_d_total[m_batch_activation_size];
	m_d_bias = &m_d_weights[m_weights_size];

	m_bias = &m_network[m_weights_size];
}
void CudaNetwork::initialize_test_data(size_t test_size) {
	size_t size = 0;

	m_test_activation_size = 0;

	for (size_t i = 1; i < m_dimensions.size(); i++) {
		size += 2 * (m_dimensions[i] * test_size);

		m_test_activation_size += m_dimensions[i] * test_size;
	}

	hipMalloc(&m_test_data, size * sizeof(float));
	m_test_activation = &m_test_data[m_test_activation_size];
}

void CudaNetwork::define(std::vector<size_t> dimensions) {
	this->m_dimensions = dimensions;

	m_weights_size = 0;
	m_bias_size = 0;
	for (size_t i = 0; i < dimensions.size() - 1; i++) {
		m_weights_size += dimensions[i] * dimensions[i + 1];
		m_bias_size += dimensions[i + 1];
	}
	m_network_size = m_weights_size + m_bias_size;
}
void CudaNetwork::compile(CudaNetwork::weight_init init) {

	float* net = (float*)calloc(m_network_size, sizeof(float));

	std::random_device rd;
	std::default_random_engine gen(rd());

	int idx = 0;
	switch (init) {
	case weight_init::xavier: {
		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {
			float lower_rand = -(1.0f / std::sqrt(m_dimensions[i + 1]));
			float upper_rand = 1.0f / std::sqrt(m_dimensions[i + 1]);

			std::uniform_real_distribution<float> dist_x(lower_rand, upper_rand);

			for (size_t j = 0; j < m_dimensions[i] * m_dimensions[i + 1]; j++, idx++) {
				net[idx] = dist_x(gen);
			}
		}
		break;
	}
	case weight_init::he: {
		float lower_rand = 0.0f;

		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {
			float upper_rand = std::sqrt(2.0f / m_dimensions[i + 1]);

			std::normal_distribution<float> dist_h(lower_rand, upper_rand);

			for (size_t j = 0; j < m_dimensions[i] * m_dimensions[i + 1]; j++, idx++) {
				net[idx] = dist_h(gen);
			}
		}
		break;
	}
	case weight_init::normalize: {
		float lower_rand = -0.5f;
		float upper_rand = 0.5f;

		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {
			std::uniform_real_distribution<float> dist_n(lower_rand, upper_rand);

			for (size_t j = 0; j < m_dimensions[i] * m_dimensions[i + 1]; j++, idx++) {
				net[idx] = dist_n(gen) * std::sqrt(1.0f / m_dimensions[i + 1]);
			}
		}
		break;
	}
	}

	hipMalloc(&m_network, m_network_size * sizeof(float));
	hipMemcpy(m_network, net, m_network_size * sizeof(float), hipMemcpyHostToDevice);

	free(net);

	std::cout << "net compiled\n";
}
 
void CudaNetwork::fit(float* x_train, float* y_train, float* x_valid, float* y_valid, size_t train_samples, size_t test_samples, size_t batch_size, size_t epochs, float learning_rate, bool shuffle, int validation_freq) {
	std::cout << "Status: network_training\n";

	auto start_time = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> time;

	const size_t iterations = train_samples / batch_size;

	float* d_x_train;
	float* d_y_train;

	float* d_x_valid;
	float* d_y_valid;

	initialize_batch_data(batch_size);
	initialize_test_data(test_samples);

	// initialize training data on the gpu
	hipMalloc(&d_x_train, train_samples * m_dimensions[0] * sizeof(float));
	hipMalloc(&d_y_train, train_samples * sizeof(float));

	hipMalloc(&d_x_valid, test_samples * m_dimensions[0] * sizeof(float));
	hipMalloc(&d_y_valid, test_samples * sizeof(float));


	hipMemcpy(d_x_train, x_train, train_samples * m_dimensions[0] * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y_train, y_train, train_samples * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(d_x_valid, x_valid, test_samples * m_dimensions[0] * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y_valid, y_valid, test_samples * sizeof(float), hipMemcpyHostToDevice);


	for (size_t e = 0; e < epochs; e++) {
		auto epoch_start_time = std::chrono::high_resolution_clock::now();

		for (size_t i = 0; i < iterations; i++) {

			float* x = &d_x_train[(i * batch_size) * m_dimensions[0]];
			float* y = &d_y_train[(i * batch_size) * m_dimensions[0]];

			forward_prop(x, m_batch_data, m_batch_activation_size, batch_size);
			back_prop(x, y, learning_rate, batch_size);
		}

		std::string tmp = "Epoch: " + std::to_string(e).append(" Time: "); int tmp_len = tmp.length();
		if (e % validation_freq == 0) {
			tmp.append(test_network(d_x_valid, d_y_valid, test_samples));
		}
		time = std::chrono::high_resolution_clock::now() - epoch_start_time;
		std::cout << tmp.insert(tmp_len, clean_time(time.count()).append(" ")).append("\n");
	}
	time = std::chrono::high_resolution_clock::now() - start_time;


	hipFree(m_batch_data);
	hipFree(m_test_data);

	hipFree(d_x_train);
	hipFree(d_y_train);
	hipFree(d_x_valid);
	hipFree(d_y_train);

	std::cout << "Status: training_complete\n";
}

void CudaNetwork::forward_prop(float* x_data, float* result_data, size_t activation_size, size_t num_elements) {

	int weight_idx = 0;
	int bias_idx = 0;

	int input_idx = 0;
	int output_idx = 0;

	for (size_t i = 0; i < m_dimensions.size() - 1; i++) {

		dim3 w_grid = (ceil(m_dimensions[i + 1] / 8), ceil(num_elements / 8), 1);
		dim3 b_grid = (ceil(m_dimensions[i + 1] / 8), 1, 1);

		float* weights = &m_network[weight_idx];
		float* bias = &m_bias[bias_idx];

		float* input = i == 0 ? &x_data[0] : &result_data[input_idx + activation_size];
		float* output = &result_data[output_idx];


		i == 0 ?
			dot_prod_t_b << < w_grid, (8, 8) >> > (weights, input, output, m_dimensions[i + 1], m_dimensions[i], num_elements, m_dimensions[i]) :
			dot_prod << < w_grid, (8, 8) >> > (weights, input, output, m_dimensions[i + 1], m_dimensions[i], m_dimensions[i], num_elements);
		hipDeviceSynchronize();

		// add bias
		horizontal_add << <ceil(m_dimensions[i + 1] / 8), 8 >> > (output, bias, m_dimensions[i + 1], num_elements);
		hipDeviceSynchronize();

		// activation
		leaky_relu << <w_grid, (8, 8) >> > (output, &output[activation_size], m_dimensions[i + 1], num_elements);
		hipDeviceSynchronize();

		weight_idx += m_dimensions[i] * m_dimensions[i + 1];
		bias_idx += m_dimensions[i + 1];

		input_idx += i == 0 ? 0 : m_dimensions[i] * num_elements;
		output_idx += m_dimensions[i + 1] * num_elements;
	}
}
void CudaNetwork::back_prop(float* x_data, float* y_data, float learning_rate, size_t num_elements) {

	const float factor = learning_rate / (float)num_elements;


	// -> compute loss
	{
		float* last_d_total = &m_d_total[m_batch_activation_size - (m_dimensions.back() * num_elements)];
		float* last_activation = &m_activation[m_batch_activation_size - (m_dimensions.back() * num_elements)];

		one_hot_loss <<< ceil(num_elements / 8), 8 >> >(last_d_total, last_activation, y_data, m_dimensions.back(), num_elements);
		hipDeviceSynchronize();
	}


	// -> compute d_total
	{
		int weight_idx = m_weights_size - (m_dimensions.back() * m_dimensions[m_dimensions.size() - 2]);
		int d_total_idx = m_batch_activation_size - (m_dimensions.back() * num_elements);

		for (size_t i = m_dimensions.size() - 2; i > 0; i--) {

			float* weight = &m_network[weight_idx];
			float* prev_total = &m_batch_data[d_total_idx - (m_dimensions[i] * num_elements)];

			float* cur_d_total = &m_d_total[d_total_idx];
			float* prev_d_total = &m_d_total[d_total_idx - (m_dimensions[i] * num_elements)];

			dot_prod_t_a << <(ceil(m_dimensions[i + 1] / 8), ceil(num_elements / 8)), (8, 8) >> > (weight, cur_d_total, prev_d_total, m_dimensions[i + 1], m_dimensions[i], m_dimensions[i + 1], num_elements);
			hipDeviceSynchronize();

			// multiply by activation function derivative
			leaky_relu_derivative << <(ceil(m_dimensions[i + 1] / 8), ceil(num_elements / 8)), (8, 8) >> > (prev_total, prev_d_total, m_dimensions[i], num_elements);
			hipDeviceSynchronize();

			d_total_idx -= m_dimensions[i] * num_elements;
			weight_idx -= m_dimensions[i] * m_dimensions[i - 1];
		}
	}

	
	// -> compute d_weights and d_biases
	{
		int activation_idx = 0;

		int d_total_idx = 0;
		int d_weights_idx = 0;
		int d_bias_idx = 0;

		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {
			dim3 w_grid(ceil(m_dimensions[i + 1] / 8), ceil(m_dimensions[i] / 8), 1);

			float* prev_activation = i == 0 ? &x_data[0] : &m_activation[activation_idx];

			float* d_total = &m_d_total[d_total_idx];
			float* d_weights = &m_d_weights[d_weights_idx];
			float* d_bias = &m_d_bias[d_bias_idx];

			// d_weights
			i == 0 ?
				dot_prod << < w_grid, (8, 8) >> > (d_total, prev_activation, d_weights, m_dimensions[i + 1], num_elements, num_elements, m_dimensions[i]) :
				dot_prod_t_b << < w_grid, (8, 8) >> > (d_total, prev_activation, d_weights, m_dimensions[i + 1], num_elements, m_dimensions[i], num_elements);
			hipDeviceSynchronize();

			// d_biases
			horizontal_sum << < ceil(m_dimensions[i + 1] / 8), 8 >> > (d_total, d_bias, m_dimensions[i + 1], num_elements);
			hipDeviceSynchronize();

			d_bias_idx += m_dimensions[i + 1];
			d_total_idx += m_dimensions[i + 1] * num_elements;
			d_weights_idx += m_dimensions[i] * m_dimensions[i + 1];
			activation_idx += i == 0 ? 0 : (m_dimensions[i] * num_elements);
		}
	}

	// update weights and biases
	{
		update_weights << < ceil(m_weights_size / 8), 8 >> > (m_network, m_d_weights, factor, m_weights_size);
		update_bias << < ceil(m_bias_size / 8), 8 >> > (m_bias, m_d_bias, factor, m_bias_size);
	}
}

std::string CudaNetwork::test_network(float* x, float* y, size_t test_size) {

	forward_prop(x, m_test_data, m_test_activation_size, test_size);

	int* d_correct;
	int correct;

	hipMalloc(&d_correct, sizeof(int));

	accuracy_score << <(ceil(test_size / 8)), 8 >> > (&m_test_activation[m_test_activation_size - (m_dimensions.back() * test_size)], y, d_correct, m_dimensions.back(), test_size);
	hipDeviceSynchronize();

	hipMemcpy(&correct, d_correct, sizeof(int), hipMemcpyDeviceToHost);

	return "score: " + std::to_string((float)correct / (float)test_size * 100.0f);
}

std::string CudaNetwork::clean_time(double time) {
	const double hour = 3600000.00;
	const double minute = 60000.00;
	const double second = 1000.00;
	std::string out;

	if (time / hour > 1.00) {
		out = std::to_string(time / hour).append(" hours");
	} else if (time / minute > 1.00) {
		out = std::to_string(time / minute).append(" minutes");
	} else if (time / second > 1.00) {
		out = std::to_string(time / second).append(" seconds");
	} else {
		out = std::to_string(time).append("(ms)");
	}

	return out;
}