#include "hip/hip_runtime.h"
#include "SingleBlockCudaNetwork.h"

#include "CudaNetActivations.cu"
#include "CudaNetDotProds.cu"

__global__ void CudaNetwork::horizontal_add(float* a, float* b, size_t a_r, size_t a_c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < a_r) {
		for (size_t j = 0; j < a_c; j++) {
			a[i * a_r + j] += b[i];
		}
	}
}
__global__ void CudaNetwork::horizontal_sum(float* a, float* b, size_t a_r, size_t a_c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < a_r) {
		b[i] = a[i * a_c];

		for (size_t j = 1; j < a_c; j++) {
			b[i] += a[i * a_c + j];
		}
	}
}

__global__ void CudaNetwork::update_weights(float* a, float* b, float lr, size_t n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n) {
		b[i] -= a[i] * lr;
	}
}
__global__ void CudaNetwork::update_bias(float* a, float* b, float lr, size_t n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n) {
		b[i] -= a[i] * lr;
	}
}

__global__ void CudaNetwork::log_loss(float* a, float* b, float* y, size_t a_r, size_t a_c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < a_c) {
		for (int j = 0; j < a_r; j++) {
			b[i * a_r + j] = a[i * a_r + j];
		}
		b[i * a_r + (int)y[i]]--;
	}
}

void CudaNetwork::fit(float* x_train, float* y_train, float* x_valid, float* y_valid, int num_elements, int batch_size, int epochs, float learning_rate, bool shuffle, int validation_freq) {
	std::cout << "Status: network_training\n";

	auto start_time = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> time;

	const int iterations = num_elements / batch_size;

	float* d_x_train;
	float* d_y_train;

	float* d_x_valid;
	float* d_y_valid;




	for (int e = 0; e < epochs; e++) {
		auto epoch_start_time = std::chrono::high_resolution_clock::now();

		for (int i = 0; i < iterations; i++) {

			float* x = &d_x_train[(i * batch_size)*];



		}

		std::string tmp = "Epoch: " + std::to_string(e).append("Time: "); int tmp_len = tmp.length();
		if (e % validation_freq == 0) {
			tmp.append(test_network(nullptr, nullptr, 0));
		}
		time = std::chrono::high_resolution_clock::now() - epoch_start_time;
		std::cout << tmp.insert(tmp_len - 1, clean_time(time.count()).append("\n"));
	}

	time = std::chrono::high_resolution_clock::now() - start_time;
	std::cout << "Status: training_complete\nAverage Epoch: " << clean_time(time.count() / (double)epochs);


}

void CudaNetwork::forward_prop(float* x_data, float* result_data, int activation_size, int num_elements) {

	const dim3 w_block(8, 8, 1);
	const dim3 b_block(8, 1, 1);
	dim3 w_grid;
	dim3 b_grid;

	int weight_idx = 0;
	int bias_idx = 0;

	int input_idx = 0;
	int output_idx = 0;

	for (size_t i = 0; i < m_dimensions.size() - 1; i++) {

		w_grid = (ceil(m_dimensions[i + 1] / 8), ceil(num_elements / 8), 1);
		b_grid = (ceil(m_dimensions[i + 1] / 8), 1, 1);

		float* weights = &m_network[weight_idx];
		float* bias = &m_bias[bias_idx];

		float* input = i == 0 ? &x_data[0] : &result_data[input_idx + activation_size];
		float* output = &result_data[output_idx];


		i == 0 ?
			dot_prod_t_b << < w_grid, w_block >> > (weights, input, output, m_dimensions[i + 1], m_dimensions[i], num_elements, m_dimensions[i]) :
			dot_prod << < w_grid, w_block >> > (weights, input, output, m_dimensions[i + 1], m_dimensions[i], m_dimensions[i], num_elements);

		// add bias
		horizontal_add << <b_grid, b_block >> > (output, bias, m_dimensions[i + 1], num_elements);

		// activation
		leaky_relu << <w_grid, w_block >> > (output, &output[activation_size], m_dimensions[i + 1], num_elements);

		weight_idx += m_dimensions[i] * m_dimensions[i + 1];
		bias_idx += m_dimensions[i + 1];

		input_idx += i == 0 ? 0 : m_dimensions[i] * num_elements;
		output_idx += m_dimensions[i + 1] * num_elements;
	}
}
void CudaNetwork::back_prop(float* x_data, float* y_data, float learning_rate, int num_elements) {

	// -> compute loss
	{
		dim3 block(8, 1, 1);
		dim3 grid(ceil(num_elements / 8), 1, 1);

		float* last_d_total = &m_d_total[m_batch_activation_size - (m_dimensions.back() * num_elements)];
		float* last_activation = &m_activation[m_batch_activation_size - (m_dimensions.back() * num_elements)];

		log_loss(last_d_total, last_activation, y_data, m_dimensions.back(), num_elements);
	}

	// -> compute d_total
	{
		dim3 block(8, 8, 1);

		int weight_idx = m_weights_size - (m_dimensions.back() * m_dimensions[m_dimensions.size() - 2]);
		int d_total_idx = m_batch_activation_size - (m_dimensions.back() * num_elements);

		for (size_t i = m_dimensions.size() - 2; i > 0; i--) {
			dim3 grid(ceil(m_dimensions[i + 1] / 8), ceil(num_elements / 8), 1);

			float* weight = &m_network[weight_idx];
			float* prev_total = &m_batch_data[d_total_idx - (m_dimensions[i] * num_elements)];

			float* cur_d_total = &m_d_total[d_total_idx];
			float* prev_d_total = &m_d_total[d_total_idx - (m_dimensions[i] * num_elements)];


			dot_prod_t_a << <grid, block >> > (weight, cur_d_total, prev_d_total, m_dimensions[i + 1], m_dimensions[i], m_dimensions[i + 1], num_elements);

			// add biases

			// multiply by activation function derivative
			leaky_relu_derivative << <grid, block >> > (prev_total, prev_d_total, m_dimensions[i], num_elements);

			d_total_idx -= m_dimensions[i] * num_elements;
			weight_idx -= m_dimensions[i] * m_dimensions[i - 1];
		}
	}

	
	// -> compute d_weights and d_biases
	{
		dim3 w_block(8, 8, 1);
		dim3 b_block(8, 1, 1);

		int activation_idx = 0;

		int d_total_idx = 0;
		int d_weights_idx = 0;
		int d_bias_idx = 0;


		for (size_t i = 0; i < m_dimensions.size() - 1; i++) {
			dim3 w_grid(ceil(m_dimensions[i + 1] / 8), ceil(m_dimensions[i] / 8), 1);
			dim3 b_grid(ceil(m_dimensions[i + 1] / 8), 1, 1);

			float* prev_activation = i == 0 ? &x_data[0] : &m_activation[activation_idx];

			float* d_total = &m_d_total[d_total_idx];
			float* d_weights = &m_d_weights[d_weights_idx];
			float* d_bias = &m_d_bias[d_bias_idx];

			// d_weights
			i == 0 ?
				dot_prod << < w_grid, w_block >> > (d_total, prev_activation, d_weights, m_dimensions[i + 1], num_elements, num_elements, m_dimensions[i]) :
				dot_prod_t_b << < w_grid, w_block >> > (d_total, prev_activation, d_weights, m_dimensions[i + 1], num_elements, m_dimensions[i], num_elements);

			// d_biases
			horizontal_sum << < b_grid, b_block >> > (d_total, d_bias, m_dimensions[i + 1], num_elements);

			d_bias_idx += m_dimensions[i + 1];
			d_total_idx += m_dimensions[i + 1] * num_elements;
			d_weights_idx += m_dimensions[i] * m_dimensions[i + 1];
			activation_idx += i == 0 ? 0 : (m_dimensions[i] * num_elements);
		}
	}

	// update weights and biases
	{
		dim3 block(8, 1, 1);
		dim3 grid(ceil(m_weights_size / 8), 1, 1);

		update_weights << < grid, block >> > (m_d_weights, m_network, m_weights_size);

		grid = (ceil(m_bias_size / 8), 1, 1);
		update_bias << < grid, block >> > (m_d_bias, m_bias, m_bias_size);
	}
}

std::string CudaNetwork::clean_time(double time) {
	const double hour = 3600000.00;
	const double minute = 60000.00;
	const double second = 1000.00;
	std::string out;

	if (time / hour > 1.00) {
		out = std::to_string(time / hour).append(" hours");
	} else if (time / minute > 1.00) {
		out = std::to_string(time / minute).append(" minutes");
	} else if (time / second > 1.00) {
		out = std::to_string(time / second).append(" seconds");
	} else {
		out = std::to_string(time).append("(ms)");
	}

	return out;
}